#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdint.h>
#include <cuda_nv12_remap_norm.h>
#include <iostream>
#include <chrono>

static constexpr uint8_t BASE_OFFSET_BITS = 7;
static constexpr uint8_t EXTRA_OFFSET_BITS = 8;
static constexpr uint8_t OFFSET_BITS = BASE_OFFSET_BITS + EXTRA_OFFSET_BITS; // 15
static constexpr int32_t OFFSET = static_cast<int32_t>(1 << BASE_OFFSET_BITS); // 128
static constexpr float SCALE  = static_cast<float>(1 << OFFSET_BITS); // 32768
static constexpr float SCALE_INV  = 1 / SCALE; // 1/32768

/*
YUV to RGB 参考BT601标准, full range：公式如下：
R = Y + 1.402 * (V - 128)
G = Y − 0.34414 * (U − 128)− 0.71414 * ( V − 128)
B = Y + 1.772 * (U − 128)
*/
// static constexpr int32_t Y_COEFF  = static_cast<int32_t>(1.0 * SCALE); // 32768
// static constexpr int32_t OY       = 0;
// static constexpr float   UV_COEFF = 1.0;
// static constexpr int32_t R_COEFF  = static_cast<int32_t>(1.402 * SCALE); // 45940
// static constexpr int32_t G_COEFF1 = static_cast<int32_t>(0.34413 * SCALE); // 11276
// static constexpr int32_t G_COEFF2 = static_cast<int32_t>(0.71414 * SCALE); // 23400
// static constexpr int32_t B_COEFF  = static_cast<int32_t>(1.772 * SCALE); // 58064

/*
YUV to RGB 参考BT601标准, limited range：公式如下：
R = (255 / 219) * (Y-16) + (255/224) * 1.402 * (V - 128)
G = (255 / 219) * (Y-16) − (255/224) * 0.34414 * (U − 128) − (255/224) * 0.71414 * (V−128)
B = (255 / 219) * (Y-16) + (255/224) * 1.772 * (U − 128)
*/
static constexpr int32_t Y_COEFF  = static_cast<int32_t>(255.0 / 219.0 * SCALE); // 38154
static constexpr int32_t OY       = 16;
static constexpr float   UV_COEFF = 255.0 / 224.0;
static constexpr int32_t R_COEFF  = static_cast<int32_t>(UV_COEFF * 1.402 * SCALE); // 52298
static constexpr int32_t G_COEFF1 = static_cast<int32_t>(UV_COEFF * 0.34413 * SCALE); // 12837
static constexpr int32_t G_COEFF2 = static_cast<int32_t>(UV_COEFF * 0.71414 * SCALE); // 26639
static constexpr int32_t B_COEFF  = static_cast<int32_t>(UV_COEFF * 1.772 * SCALE); // 66100

/*
YUV to RGB 参考BT709标准, full range：公式如下：
R = Y + 1.5748 * (V - 128)
G = Y − 0.1868 * (U − 128) − 0.4680 * ( V − 128)
B = Y + 1.856  * (U − 128)
*/
// static constexpr int32_t Y_COEFF  = static_cast<int32_t>(1.0 * SCALE); // 32768
// static constexpr int32_t OY       = 0;
// static constexpr float   UV_COEFF = 1;
// static constexpr int32_t R_COEFF  = static_cast<int32_t>(UV_COEFF * 1.5748 * SCALE); // 51603
// static constexpr int32_t G_COEFF1 = static_cast<int32_t>(UV_COEFF * 0.1868 * SCALE); // 6121
// static constexpr int32_t G_COEFF2 = static_cast<int32_t>(UV_COEFF * 0.4680 * SCALE); // 15335
// static constexpr int32_t B_COEFF  = static_cast<int32_t>(UV_COEFF * 1.856 * SCALE); // 60817
template<typename _T>
static __host__ __device__ __forceinline__ uint8_t u8cast(_T value){
    return value < 0 ? 0 : (value >= 255 ? 255 : value);
}

template <typename T>
__global__ void REMAP_TO_RGB8_GRAY_CUDA(const uint8_t* src, const int16_t *map_xy, const float* mean,
  const float* std_inv, const float* max_v, int src_width, int src_height, int dst_width, int dst_height, uint8_t* rgb, uint8_t* gray, YUVType type) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int batch = blockIdx.z;
    const int src_size = src_width * src_height;
    const int dst_size = dst_width * dst_height;
    const int dst_batch_offset = batch * dst_size;
    const int src_batch_offset = batch * (type == NV12 ? (src_size * 3 / 2) : (src_size << 1));

    if (x < dst_width && y < dst_height) {
        int index_dst = y * dst_width + x;

        int16_t index_src_x = map_xy[(index_dst + dst_batch_offset)*2];
        int16_t index_src_y = map_xy[(index_dst + dst_batch_offset)*2 + 1];

        if (index_src_x < 0 || index_src_y < 0 || index_src_x >= src_width || index_src_y >= src_height) {
            rgb[index_dst + dst_batch_offset*3] = u8cast( (0 - mean[0]) * std_inv[0] ); // B
            rgb[index_dst + dst_batch_offset*3 + dst_size] = u8cast((0 - mean[1]) * std_inv[1]) ; // G
            rgb[index_dst + dst_batch_offset*3 + dst_size*2] = u8cast((0 - mean[2]) * std_inv[2]); // R
            if (gray != nullptr) {
                gray[index_dst + dst_batch_offset] = 0;
            }
        } else {
            int index_src;
            uint8_t Y, U, V;

            if (type == NV12) {
                index_src = index_src_y * src_width + index_src_x + src_batch_offset;
                int index_src_uv = (index_src_y / 2) * src_width + (index_src_x / 2) * 2 + src_batch_offset;
                Y = src[index_src];
                U = src[src_size + index_src_uv];
                V = src[src_size + index_src_uv + 1];
            } else {
                index_src = index_src_y * src_width * 2 + index_src_x * 2 + src_batch_offset;
                Y = src[index_src];
                U = src[index_src_x % 2 == 0 ? index_src + 1 : index_src - 1];
                V = src[index_src_x % 2 == 0 ? index_src + 3 : index_src + 1];
            }

            if (gray != nullptr) {
                gray[index_dst + dst_batch_offset] = Y;
            }

            int32_t Y_16 = static_cast<int32_t>(Y - OY);
            T tmp_R = static_cast<T>((Y_COEFF * Y_16 + (R_COEFF * (static_cast<int32_t>(V) - OFFSET))));
            T tmp_G = static_cast<T>((Y_COEFF * Y_16 - (G_COEFF1 * (static_cast<int32_t>(U) - OFFSET) + G_COEFF2 * (static_cast<int32_t>(V) - OFFSET))));
            T tmp_B = static_cast<T>((Y_COEFF * Y_16 + (B_COEFF * (static_cast<int32_t>(U) - OFFSET))));

            // 饱和截断处理
            tmp_R = tmp_R < 0 ? 0 : (tmp_R > (*max_v) ? (*max_v) : tmp_R);
            tmp_G = tmp_G < 0 ? 0 : (tmp_G > (*max_v) ? (*max_v) : tmp_G);
            tmp_B = tmp_B < 0 ? 0 : (tmp_B > (*max_v) ? (*max_v) : tmp_B);

            index_dst += dst_batch_offset*3;
            //rgb[index_dst] = (tmp_B - mean[0]) * std_inv[0];
            //rgb[index_dst + dst_size] = (tmp_G - mean[1]) * std_inv[1];
            //rgb[index_dst + dst_size*2] = (tmp_R - mean[2]) * std_inv[2];
            // for show rgb picture
            rgb[index_dst*3 + dst_batch_offset*3] = u8cast((tmp_R - mean[0]) * std_inv[0]);;
            rgb[index_dst* 3+1 + dst_batch_offset*3] = u8cast((tmp_G - mean[1]) * std_inv[1]);
            rgb[index_dst* 3+2 + dst_batch_offset*3] = u8cast((tmp_B - mean[2]) * std_inv[2]);
        }
    }
}

void NV12ToYUV422(const void* nv12, int width, int height, void* yuv422) {
    const uint8_t* src = (const uint8_t*)nv12;
    uint8_t* dst = (uint8_t*)yuv422;

    int frame_size = width * height;
    int uv_offset = frame_size;

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x += 2) {
            // Y values
            uint8_t Y1 = src[y * width + x];
            uint8_t Y2 = src[y * width + x + 1];

            // UV values (NV12 has interleaved UV data for each 2x2 block)
            int uv_index = (y / 2) * width + x;
            uint8_t U = src[uv_offset + uv_index];
            uint8_t V = src[uv_offset + uv_index + 1];

            // YUV422 stores data in YUVY format for 2 pixels
            int output_index = (y * width + x) * 2;
            dst[output_index] = Y1;       // Y1
            dst[output_index + 1] = U;    // U
            dst[output_index + 2] = Y2;   // Y2
            dst[output_index + 3] = V;    // V
        }
    }
}

extern "C" int32_t YUVRemapAndSplit(YUVRemapAndSplitParameters& param, const void* yuv_image, const void* map_xy,
  void *gray, void *image_nchw, void* workspace, hipStream_t stream, YUVType yuv_type)
{
  // dim3 block_dim(16, 16);
  dim3 block_dim(128, 1);
  dim3 grid_dim((param.output_w + block_dim.x - 1) / block_dim.x, (param.output_h + block_dim.y - 1) / block_dim.y, param.batchSize);
  uint8_t *workspace_ = (uint8_t *)workspace;
  float *mean_ = (float *)workspace_;
  workspace_ += 4 * sizeof(float);
  float *std_inv_ = (float *)workspace_;
  workspace_ += 4 * sizeof(float);
  float *max_v_ = (float *)workspace_;
  workspace_ += 1 * sizeof(float);

#ifdef __aarch64__
  float *mean_stdinv_float_t = mean_;
  float *stdinv_stdinv_float = std_inv_;
  float *max_v = max_v_;
#else
  float mean_stdinv_float_t[4];
  float stdinv_stdinv_float[4];
  float max_v[1];
#endif
  mean_stdinv_float_t[0] = static_cast<float>(param.mean[0]) * SCALE;
  mean_stdinv_float_t[1] = static_cast<float>(param.mean[1]) * SCALE;
  mean_stdinv_float_t[2] = static_cast<float>(param.mean[2]) * SCALE;

  stdinv_stdinv_float[0] = static_cast<float>(param.std_inv[0]) * SCALE_INV;
  stdinv_stdinv_float[1] = static_cast<float>(param.std_inv[1]) * SCALE_INV;
  stdinv_stdinv_float[2] = static_cast<float>(param.std_inv[2]) * SCALE_INV;
  max_v[0] = static_cast<float>(UINT8_MAX * SCALE);

#ifdef __aarch64__
  // Nothing
  // memcpy(mean_, &mean_stdinv_int16_t[0], 8*sizeof(int16_t));
#else
  hipMemcpyAsync(mean_, &mean_stdinv_float_t[0], 4*sizeof(float), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(std_inv_, &stdinv_stdinv_float[0], 4*sizeof(float), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(max_v_, &max_v[0], 1 * sizeof(float), hipMemcpyHostToDevice, stream);
#endif
  // hipMemcpy(std_inv_, &std_inv_int16[0], 3*sizeof(int16_t), hipMemcpyHostToDevice);
#if 1
  // 检查常量
  // printf("%d\n", UINT8_MAX);
  // printf("%d\n", BASE_OFFSET_BITS);
  // printf("%d\n", EXTRA_OFFSET_BITS);
  // printf("%d\n", OFFSET_BITS);

  // printf("%d\n", Y_COEFF);
  // printf("%d\n", OY);
  // printf("%f\n", UV_COEFF);
  // printf("%d\n", OFFSET);
  // printf("%f\n", SCALE);
  // printf("%f\n", SCALE_INV);
  // printf("%d\n", R_COEFF);
  // printf("%d\n", G_COEFF1);
  // printf("%d\n", G_COEFF2);
  // printf("%d\n", B_COEFF);
  // printf("%f\n", *max_v);
  REMAP_TO_RGB8_GRAY_CUDA<float><<<grid_dim, block_dim, 0, stream>>>((const uint8_t *)yuv_image, (const int16_t *)map_xy, mean_, std_inv_, max_v_,
    param.intput_w, param.intput_h, param.output_w, param.output_h, (uint8_t *)image_nchw, (uint8_t *)gray, yuv_type);

#else
  constexpr int32_t test_count = 100;
  auto time_2 = std::chrono::steady_clock::now();
  for (int i = 0; i < test_count; ++i) {
    // Launch the CUDA kernel
    NV12_REMAP_TO_RGB8_GRAY_CUDA<float><<<grid_dim, block_dim, 0, stream>>>((const uint8_t *)yuv_image, (const int16_t *)map_xy, mean_, std_inv_,
      param.intput_w, param.intput_h, param.output_w, param.output_h, (float *)image_nchw, (uint8_t *)gray);
  }

  // 同步CUDA流
  hipStreamSynchronize(stream);

  auto time_3 = std::chrono::steady_clock::now();
  auto time_d1 = std::chrono::duration_cast<std::chrono::microseconds>((time_3 - time_2)).count();
  std::cout << "cuda remap + convert: " << time_d1 / 1000.0 / test_count << " ms" << std::endl;
#endif
  hipError_t status = hipGetLastError();
  if (!status) {
    return status;
  }

  return 0;
}
